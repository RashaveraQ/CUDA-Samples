#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes

#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__global__ void
cudaProcess(unsigned int *g_odata, int *dst, int *src, int WIDTH, int HEIGHT, int mouse_buttons, int mouse_x, int mouse_y)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

	if (x <= 0 || WIDTH <= x || y <= 0 || HEIGHT <= y)
		return;

	int s;
	switch (mouse_buttons) {
	case 1:
		s = (x == mouse_x && y == mouse_y) ? 3 : 2;
		break;
	case 4:
		s = (x == mouse_x && y == mouse_y) ? 4 : 2;
		break;
	default:
		s = src[(y - 1) * WIDTH + (x - 1)] + src[(y - 1) * WIDTH + x] + src[(y - 1) * WIDTH + (x + 1)]
			+ src[y * WIDTH + (x - 1)] + src[y * WIDTH + (x + 1)]
			+ src[(y + 1) * WIDTH + (x - 1)] + src[(y + 1) * WIDTH + x] + src[(y + 1) * WIDTH + (x + 1)];
		break;
	}

	switch (s) {
	case 2:	// �ێ�
		dst[y * WIDTH + x] = src[y * WIDTH + x];
		break;
	case 3:	// �a��
		dst[y * WIDTH + x] = 1;
		break;
	default: // ����
		dst[y * WIDTH + x] = 0;
		break;
	}

    uchar4 c4 = (dst[y * WIDTH + x] == 1) ? make_uchar4(255,255,255,0) : make_uchar4(0,0,0,0);
	g_odata[y * WIDTH + x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void
launch_cudaProcess(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int *d_dst, int *d_src, int WIDTH, int HEIGHT, int mouse_buttons, int mouse_x, int mouse_y)
{
    cudaProcess<<< grid, block, sbytes >>>(g_odata, d_dst, d_src, WIDTH, HEIGHT, mouse_buttons, mouse_x, mouse_y);
}

__global__ void
cudaProcess_setPixel(unsigned int *g_odata, int imgw, int x, int y, bool set)
{
	uchar4 c4 = set ? make_uchar4(255, 255, 0, 0) : make_uchar4(20, 20, 20, 0);
	g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void
launch_cudaProcess_setPixel(unsigned int *g_odata, int imgw, int x, int y, bool set)
{
	cudaProcess_setPixel <<<1, 0 >>>(g_odata, imgw, x, y, set);
}