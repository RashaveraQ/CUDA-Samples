
#include <hip/hip_runtime.h>
__device__ static unsigned long xors_x = 123456789;
__device__ static unsigned long xors_y = 362436069;
__device__ static unsigned long xors_z = 521288629;
__device__ static unsigned long xors_w = 88675123;
 
__device__
unsigned long Xorshift128()
{ 
    unsigned long t; 
    t = (xors_x^(xors_x<<11));
    xors_x = xors_y; xors_y = xors_z; xors_z = xors_w; 
    return ( xors_w = (xors_w^(xors_w>>19))^(t^(t>>8)) ); 
}
__device__
long Xorshift128(long l, long h)
{ 
    unsigned long t; 
    t = (xors_x^(xors_x<<11));
    xors_x = xors_y; xors_y = xors_z; xors_z = xors_w; 
    xors_w = (xors_w^(xors_w>>19))^(t^(t>>8));
    return l+(xors_w%(h-l));
}
 
__device__
float XorFrand(float l, float h)
{
    return l+(h-l)*(Xorshift128(0, 1000000)/1000000.0f);
}
