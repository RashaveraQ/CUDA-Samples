#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#include "SolarWind.h"
#include <timer.h>               // timing functions

#define DEPTH 8


//const char *sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, float4* vec, float3* pAxis, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = (x / DEPTH) / (float) (width / DEPTH);
    float v = (y / DEPTH) / (float) (height / DEPTH);
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    //float freq = 4.0f;
    //float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;
	float4 p = pos[y * width + x];
	float4 V = vec[y * width + x];

	float3 n = make_float3(p.x - pAxis->x, p.y - pAxis->y, p.z - pAxis->z);
	float3 s = make_float3(p.x + pAxis->x, p.y + pAxis->y, p.z + pAxis->z);
	float  n2 = sqrtf(n.x * n.x + n.y * n.y + n.z * n.z);
	float  s2 = sqrtf(s.x * s.x + s.y * s.y + s.z * s.z);
	float  n3 = n2 * n2 * n2;
	float  s3 = s2 * s2 * s2;
	float3 B = make_float3(	n.x / n3 - s.x / s3, n.y / n3 - s.y / s3, n.z / n3 - s.z / s3);

	// calculate accelerate of particle.
	float3 A = make_float3(	V.y * B.z - V.z * B.y, V.z * B.x - V.x * B.z, V.x * B.y - V.y * B.x);

	// update velocity of particle.
	V.x += A.x;
	V.y += A.y;
	V.z += A.z;
	V.w += 0.1f;

	// update position of particle.
	p.x += V.x;
	p.y += V.y;
	p.z += V.z;

	pos[y * width + x] = p;
	vec[y * width + x] = V;

	float l = p.x * p.x + p.y * p.y + p.z * p.z;
	// check boundary
	if (V.w > 100.0f || l > 6.0f) {
		// reset a particle.
		float w = 0.4f * ((x % DEPTH) * DEPTH + (y % DEPTH)) / (DEPTH * DEPTH);
		pos[y * width + x] = make_float4(-2.0f + w, u, v, 1.0f);
		vec[y * width + x] = make_float4(0.01f /* XorFrand(0.005f,0.02f) */, 0.0f, 0.0f, 0.0f);
	}
}

void launch_kernel(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block>>>(pos, d_vec, d_axis, mesh_width, mesh_height, time);
}

bool checkHW(char *name, const char *gpuType, int dev)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    strcpy(name, deviceProp.name);

    if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
    {
        return true;
    }
    else
    {
        return false;
    }
}

int findGraphicsGPU(char *name)
{
    int nGraphicsGPU = 0;
    int deviceCount = 0;
    bool bFoundGraphics = false;
    char firstGraphicsName[256], temp[256];

    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("> FAILED %s sample finished, exiting...\n", sSDKsample);
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("> There are no device(s) supporting CUDA\n");
        return false;
    }
    else
    {
        printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
        printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

        if (bGraphics)
        {
            if (!bFoundGraphics)
            {
                strcpy(firstGraphicsName, temp);
            }

            nGraphicsGPU++;
        }
    }

    if (nGraphicsGPU)
    {
        strcpy(name, firstGraphicsName);
    }
    else
    {
        strcpy(name, "this hardware");
    }

    return nGraphicsGPU;
}

void reset()
{
	int N = mesh_width * mesh_height;
	size_t size = N * sizeof(float4);
	hipMemcpy(d_vec, h_vec, size, hipMemcpyHostToDevice);
}

void setAxis(float x, float y, float z)
{
	h_axis = make_float3(x, y, z);
	h_axis_radius = sqrtf(h_axis.x * h_axis.x + h_axis.y * h_axis.y + h_axis.z * h_axis.z);

	gkLightPos[0] = h_axis.x;
	gkLightPos[1] = h_axis.y;
	gkLightPos[2] = h_axis.z;
	gkLightPos[3] = 0;

	gkLightPos2[0] = -h_axis.x;
	gkLightPos2[1] = -h_axis.y;
	gkLightPos2[2] = -h_axis.z;
	gkLightPos2[3] = 0;

	size_t size = sizeof(float3);
	hipMalloc(&d_axis, size);
	hipMemcpy(d_axis, &h_axis, size, hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    printf("\n");

	int N = mesh_width * mesh_height;
	size_t size = N * sizeof(float4);
	h_vec = (float4*)malloc(size);
	for (int i = 0; i < N; i++) {
		h_vec[i] = make_float4(0.0f, 0.01f, 0.0f, 200.0f);
	}
	hipMalloc(&d_vec, size);

	setAxis(0.0, 0.001f, 0.0);

	reset();

    runTest(argc, argv, ref_file);

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

	hipFree(d_vec);
	free(h_vec);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    glewInit();

    if (! glewIsSupported("GL_VERSION_2_0 "))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // command line mode only
    if (ref_file != NULL)
    {
        // This will pick the best possible CUDA capable device
        int devID = findCudaDevice(argc, (const char **)argv);

        // create VBO
        checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, mesh_width*mesh_height*4*sizeof(float)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;

        // hipDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling hipDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
    }
    else
    {
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        if (checkCmdLineFlag(argc, (const char **)argv, "device"))
        {
            if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
            {
                return false;
            }
        }
        else
        {
            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }

        // register callbacks
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

        // create VBO
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // run the cuda part
        runCuda(&cuda_vbo_resource);

        // start rendering mainloop
        glutMainLoop();

    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(mesh_width*mesh_height*sizeof(float));

    // execute the kernel
    launch_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width*mesh_height*sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                mesh_width*mesh_height*sizeof(float),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}



void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            exit(EXIT_SUCCESS);
            break;

		case 'x':
			setAxis(0.001f, 0.0, 0.0);
			break;

		case 'X':
			setAxis(-0.001f, 0.0, 0.0);
			break;

		case 'y':
			setAxis(0.0, 0.001f, 0.0);
			break;

		case 'Y':
			setAxis(0.0, -0.001f, 0.0);
			break;

		case 'z':
			setAxis(0.0, 0.0, 0.001f);
			break;

		case 'Z':
			setAxis(0.0, 0.0, -0.001f);
			break;

		case 'r':
			reset();
			break;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

	switch (glutGetModifiers()) {
	default:
		if (mouse_buttons & 1)
		{
			rotate_x += dy * 0.2f;
			rotate_y += dx * 0.2f;
		}
		else if (mouse_buttons & 4)
		{
			translate_z += dy * 0.01f;
		}
		break;

	case GLUT_ACTIVE_CTRL:
		if (mouse_buttons & 1)
		{
			float th  = 0.01f * dx;
			float phy = 0.01f * dy;
			float sin_th = sin(th);
			float cos_th = cos(th);
			float sin_phy = sin(phy);
			float cos_phy = cos(phy);
			setAxis(cos_th * h_axis.x - sin_th * cos_phy * h_axis.y + sin_th * sin_phy * h_axis.z, 
			        sin_th * h_axis.x + cos_th * cos_phy * h_axis.y - cos_th * sin_phy * h_axis.z,
					                             sin_phy * h_axis.y +          cos_phy * h_axis.z );
		}
		else if (mouse_buttons & 4)
		{
			float r = (dy > 0) ? 1.05f : 0.95f;
			setAxis(r * h_axis.x, r * h_axis.y, r * h_axis.z);
		}
		break;
	}

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER_ARB, vbo);
        float *data = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<float>("./data/regression.dat",
                                data, mesh_width * mesh_height * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}
